
#include <hip/hip_runtime.h>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
			 bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/****
 sumall - sum no_imgs images in matrix sumObjs, starting from firstImg
****/
__global__
void sumall( float* sumObjs, float** array_Img, int no_imgs, int Xdim, int Ydim){

   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int offset = x+y*Xdim;
   int px_img = Xdim*Ydim;

   if ((x<Xdim)&&(y<Ydim)){
//   	printf("no_imgs=%d, (%d, %d) th=(%d,%d) blk=(%d,%d)\n", no_imgs, x, y, threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
	float sum=0;
	float* ptr;
	for (int img=0; img<no_imgs; img++){
	    	ptr=array_Img[img];
		sum+= ptr[offset];
	}
        sum*=2;
        sumObjs[x+y*Xdim]=sum;
//	printf("data %f\n", sumObjs[x+y+y*Xdim]);

   } //end-if Xdim, Ydim
} 
